
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out, float * d_in) {
int idx = threadIdx.x;
float f = d_in[idx];
d_out[idx] = f * f;
}


int main(int argc, char ** argv) {
   const int ARRAY_SIZE = 64;
   const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
   //"Declare the size of the arrange in constant ARRAY_SIZE 
   //and determine how many bytes it uses declaring constant
   //ARRAY_BYTES."

   int i;
   

   // generate the input array on the host
   // As an convention the host variables carry an initial letter h, and the 
   // device variables carry an initial letter d
   float h_in[ARRAY_SIZE];
   for (i=0; i< ARRAY_SIZE; i++) {
   h_in[i] = float(i);
   }

   float h_out[ARRAY_SIZE];

   //declare GPU memory pointers
   float * d_in;
   float * d_out;

   hipMalloc((void **) &d_in, ARRAY_BYTES);
   hipMalloc((void **) &d_out, ARRAY_BYTES);

   //transfer the array to the GPU
   hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  //lunch the kernel
   square<<<1, ARRAY_SIZE>>>(d_out, d_in);

   //copy back the result array to the CPU
   hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

   //print out the resulting array
   for (int i=0; i < ARRAY_SIZE; i++) {
      printf("%f", h_out[i]);
      printf(((i % 4) != 3) ? "\t" : "\n");
   }

   //free GPU memory allocation
   hipFree(d_in);
   hipFree(d_out);

   return 0;

}
